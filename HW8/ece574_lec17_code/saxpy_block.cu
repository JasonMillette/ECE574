/* Based on code from here: http://devblogs.nvidia.com/parallelforall/easy-introduction-cuda-c-and-c/ */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/* Calculate SAXPY, single-precision vector math 	*/
/* y[i]=a*x[i]+y[i]					*/

__global__
void saxpy (int n, float a, float *x, float *y) {

	int i=blockIdx.x*blockDim.x+threadIdx.x;

	/* Only run calculation if we are in range */
	/* where i is valid.  It can be out of range */
	/* if our vector is shorter than a */
	/* multiple of the blocksize */

	if (i<n) {
		y[i]=a*x[i]+y[i];
	}
}

int main(int argc, char **argv) {

	int i,j;
	float *x, *y, *dev_x, *dev_y;
	float a;
	long long N=(1000*1000*8),loops=1;

	if (argc>1) {
		N=atoll(argv[1]);
	}

	if (argc>2) {
		loops=atoll(argv[2]);
	}

	/* Allocate vectors on CPU */
	x=(float *)malloc(N*sizeof(float));
	y=(float *)malloc(N*sizeof(float));

	/* Allocate vectors on GPU */
	hipMalloc((void **)&dev_x,N*sizeof(float));
	hipMalloc((void **)&dev_y,N*sizeof(float));

	/* Initialize the host vectors */
	for(i=0;i<N;i++) {
		x[i]=(float)i;
		y[i]=(float)(10.0*i);
	}

	hipMemcpy(dev_x,x,N*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,N*sizeof(float),hipMemcpyHostToDevice);

	printf("Size: %d\n",(N+255)/256);

	a=5.0;

	for(j=0;j<loops;j++) {
		/* Perform SAXPY */
		saxpy<<<(N+255)/256,256>>>(N,a,dev_x,dev_y);
	}	

	// make the host block until the device is finished
	hipDeviceSynchronize();

	// check for error
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
 	}
	
	hipMemcpy(y,dev_y,N*sizeof(float),hipMemcpyDeviceToHost);

	/* results */
	i=100;
	printf("y[%d]=%f, y[%lld]=%f\n",i,y[i],N-1,y[N-1]);

	/* y[i]=a*x[i]+y[i] */
	/* 0: a=5, x=0, y=0  ::::::: y=0 */
	/* 1: a=5, x=1, y=10 ::::::: y=15 */
	/* 2: a=5, x=2, y=20 ::::::: y=30 */
	/* 3: a=5, x=3, y=30 ::::::: y=45 */
	/* 4: a=5, x=4, y=40 ::::::: y=60 */
	/* ... */
	/* 100: a=5, x=100, y=1000 y=1500 */

	hipFree(dev_x);
	hipFree(dev_y);

	return 0;
}

