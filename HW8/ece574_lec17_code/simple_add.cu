
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add (int a, int b, int *c) {
	*c=a+b;
}

int main(int argc, char **argv) {

	int c;
	int *dev_c;

	/* Allocate memory on device */
	/* Note, the pointer returned is *not* valid on the host */
	/* and dereferencing it will not work */
	hipMalloc( (void **)&dev_c,sizeof(int));

	add<<<1,1>>>(3,4,dev_c);

	hipMemcpy(	&c,
			dev_c,
			sizeof(int),
			hipMemcpyDeviceToHost);

	printf("3+4=%d\n",c);

	hipFree(dev_c);

	return 0;

}
