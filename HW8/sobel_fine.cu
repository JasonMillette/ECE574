#include "hip/hip_runtime.h"
/* Example sobel code for ECE574 -- Spring 2019 */
/* By Vince Weaver <vincent.weaver@maine.edu> */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <errno.h>
#include <math.h>

#include <jpeglib.h>

#include <hip/hip_runtime.h>

#include <papi.h>

/* Filters */

/* Structure describing the image */
struct image_t {
	int x;
	int y;
	int depth;	/* bytes */
	unsigned char *pixels;
};

struct convolve_data_t {
	struct image_t *old;
	struct image_t *newt;
	int (*filter)[3][3];
	int ystart;
	int yend;
};

//global for CUDA function
__global__
void combine(unsigned char *x, unsigned char *y, unsigned char *dev_new, int imageSize) {
	int i = blockIdx.x * blockDim.x+threadIdx.x; //ditermins the pixels to be operated on
	int out;

	if (i < imageSize) {
		out=sqrt((double)((x[i]*x[i]) + (y[i]*y[i]))); //double to make nvcc happy
		if (out>255) out=255;
		if (out<0) out=0;
		dev_new[i]=out;
	}

	return; 
}

static int load_jpeg(char *filename, struct image_t *image) {

	FILE *fff;
	struct jpeg_decompress_struct cinfo;
	struct jpeg_error_mgr jerr;
	JSAMPROW output_data;
	unsigned int scanline_len;
	int scanline_count=0;

	fff=fopen(filename,"rb");
	if (fff==NULL) {
		fprintf(stderr, "Could not load %s: %s\n",
			filename, strerror(errno));
		return -1;
	}

	/* set up jpeg error routines */
	cinfo.err = jpeg_std_error(&jerr);

	/* Initialize cinfo */
	jpeg_create_decompress(&cinfo);

	/* Set input file */
	jpeg_stdio_src(&cinfo, fff);

	/* read header */
	jpeg_read_header(&cinfo, TRUE);

	/* Start decompressor */
	jpeg_start_decompress(&cinfo);

	printf("output_width=%d, output_height=%d, output_components=%d\n",
		cinfo.output_width,
		cinfo.output_height,
		cinfo.output_components);

	image->x=cinfo.output_width;
	image->y=cinfo.output_height;
	image->depth=cinfo.output_components;

	scanline_len = cinfo.output_width * cinfo.output_components;
	image->pixels=(unsigned char *)malloc(cinfo.output_width * cinfo.output_height * cinfo.output_components);

	while (scanline_count < cinfo.output_height) {
		output_data = (image->pixels + (scanline_count * scanline_len));
		jpeg_read_scanlines(&cinfo, &output_data, 1);
		scanline_count++;
	}

	/* Finish decompressing */
	jpeg_finish_decompress(&cinfo);

	jpeg_destroy_decompress(&cinfo);

	fclose(fff);

	return 0;
}

static int store_jpeg(const char *filename, struct image_t *image) {

	struct jpeg_compress_struct cinfo;
	struct jpeg_error_mgr jerr;
	int quality=90; /* % */
	int i;

	FILE *fff;

	JSAMPROW row_pointer[1];
	int row_stride;

	/* setup error handler */
	cinfo.err = jpeg_std_error(&jerr);

	/* initialize jpeg compression object */
	jpeg_create_compress(&cinfo);

	/* Open file */
	fff = fopen(filename, "wb");
	if (fff==NULL) {
		fprintf(stderr, "can't open %s: %s\n",
			filename,strerror(errno));
		return -1;
	}

	jpeg_stdio_dest(&cinfo, fff);

	/* Set compression parameters */
	cinfo.image_width = image->x;
	cinfo.image_height = image->y;
	cinfo.input_components = image->depth;
	cinfo.in_color_space = JCS_RGB;
	jpeg_set_defaults(&cinfo);
	jpeg_set_quality(&cinfo, quality, TRUE);

	/* start compressing */
	jpeg_start_compress(&cinfo, TRUE);

	row_stride=image->x*image->depth;

	for(i=0;i<image->y;i++) {
		row_pointer[0] = & image->pixels[i * row_stride];
		jpeg_write_scanlines(&cinfo, row_pointer, 1);
	}

	/* finish compressing */
	jpeg_finish_compress(&cinfo);

	/* close file */
	fclose(fff);

	/* clean up */
	jpeg_destroy_compress(&cinfo);

	return 0;
}

__global__
void cuda_generic_convolve (int imageSize, unsigned char *in, int *matrix, unsigned char *out, int xsize, int depth) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	int sum = 0;
	if((i%(xsize*depth) >= depth) && (i%(xsize*depth) <= (xsize*depth-depth-1)) && (i >= (xsize*depth+depth)) && (i <= (imageSize-xsize*depth-depth-1)) && (i < imageSize)) {
		sum+=in[i-3-(xsize*depth)]*matrix[0];
		sum+=in[i-(xsize*depth)]*matrix[1];
		sum+=in[i+3-(xsize*depth)]*matrix[2];
		sum+=in[i-3]*matrix[3];
		sum+=in[i]*matrix[4];
		sum+=in[i+3]*matrix[5];
		sum+=in[i-3+(xsize*depth)]*matrix[6];
		sum+=in[i+(xsize*depth)]*matrix[7];
		sum+=in[i+3+(xsize*depth)]*matrix[8];

		if (sum<0) sum=0;
		if (sum>255) sum=255;

		out[i] = sum;
	}
		return;
}

int main(int argc, char **argv) {

	struct image_t image,sobel_x,sobel_y,new_image; 
	long long start_time,load_time,convolve_time;
	long long combine_after=0,combine_before=0;
	long long copy_before=0,copy_after=0,copy2_before=0,copy2_after=0;
	long long store_after,store_before;

	/* Check command line usage */
	if (argc<2) {
		fprintf(stderr,"Usage: %s image_file\n",argv[0]);
		return -1;
	}

	PAPI_library_init(PAPI_VER_CURRENT);

	start_time=PAPI_get_real_usec();

	/* Load an image */
	load_jpeg(argv[1],&image);

	load_time=PAPI_get_real_usec();

	/* Allocate space for output image */
	new_image.x=image.x;
	new_image.y=image.y;
	new_image.depth=image.depth;
	new_image.pixels=(unsigned char *)calloc(image.x*image.y*image.depth,sizeof(char));

	/* Allocate space for output image */
	sobel_x.x=image.x;
	sobel_x.y=image.y;
	sobel_x.depth=image.depth;
	sobel_x.pixels=(unsigned char *)calloc(image.x*image.y*image.depth,sizeof(char));

	/* Allocate space for output image */
	sobel_y.x=image.x;
	sobel_y.y=image.y;
	sobel_y.depth=image.depth;
	sobel_y.pixels=(unsigned char *)calloc(image.x*image.y*image.depth,sizeof(char));

	//Allocating GPU memory
	unsigned char *dev_in_y, *dev_out_x, *dev_out_y, *dev_in_x;
	int imageSize = image.y * image.x * image.depth, *dev_xFilter, *dev_yFilter;
	int xFilter[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
	int yFilter[9] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};
	hipMalloc((void **)&dev_in_y,(imageSize*sizeof(char)));
	hipMalloc((void **)&dev_out_y,(imageSize*sizeof(char)));
	hipMalloc((void **)&dev_in_x,(imageSize*sizeof(char)));
	hipMalloc((void **)&dev_out_x,(imageSize*sizeof(char)));
	hipMalloc((void **)&dev_xFilter,(9*sizeof(int)));
	hipMalloc((void **)&dev_yFilter,(9*sizeof(int)));

	//copying memory to GPU
	hipMemcpy(dev_in_y, image.pixels, (imageSize*sizeof(char)),hipMemcpyHostToDevice);
	hipMemcpy(dev_in_x, image.pixels, (imageSize*sizeof(char)),hipMemcpyHostToDevice);
	hipMemcpy(dev_yFilter, yFilter, (9*sizeof(int)),hipMemcpyHostToDevice);
	hipMemcpy(dev_xFilter, xFilter, (9*sizeof(int)),hipMemcpyHostToDevice);

	//convolving
	cuda_generic_convolve<<<(imageSize+255)/256,256>>>(imageSize, dev_in_y, dev_yFilter, dev_out_y, image.x, image.depth);
	hipDeviceSynchronize();
	cuda_generic_convolve<<<(imageSize+255)/256,256>>>(imageSize, dev_in_x, dev_xFilter, dev_out_x, image.x, image.depth);

	// wait for convolves to finish
	hipDeviceSynchronize();
	//check for errors
	hipError_t error = hipGetLastError();
	if( error != hipSuccess) {
		printf("%s Big oof!\n", hipGetErrorString(error));
	}

	hipMemcpy(sobel_y.pixels, dev_out_y, (imageSize*sizeof(char)),hipMemcpyDeviceToHost);
	hipMemcpy(sobel_x.pixels, dev_out_x, (imageSize*sizeof(char)),hipMemcpyDeviceToHost);

	convolve_time=PAPI_get_real_usec();

	/* Combine to form output */
	//Allocating memory on GPU
	unsigned char *dev_sobelx, *dev_sobely, *dev_new;

	hipMalloc((void **)&dev_sobelx,(image.x*image.y*image.depth*sizeof(char)));
	hipMalloc((void **)&dev_sobely,(image.x*image.y*image.depth*sizeof(char)));
	hipMalloc((void **)&dev_new,(image.x*image.y*image.depth*sizeof(char)));

	//copying memory to GPU
	copy_before = PAPI_get_real_usec();
	hipMemcpy(dev_sobelx, sobel_x.pixels, (imageSize*sizeof(char)),hipMemcpyHostToDevice);
	hipMemcpy(dev_sobely, sobel_y.pixels, (imageSize*sizeof(char)),hipMemcpyHostToDevice);
	copy_after= PAPI_get_real_usec();

	combine_before = PAPI_get_real_usec();
	combine<<<(imageSize+255)/256,256>>>(dev_sobelx, dev_sobely, dev_new, imageSize);

	// wait for combixe to finish
	hipDeviceSynchronize();
	combine_after= PAPI_get_real_usec();

	/* REPLACE THE ABOVE WITH YOUR CODE */
	/* IT SHOULD ALLOCATE SPACE ON DEVICE */
	/* COPY SOBEL_X and SOBEL_Y data to device */
	/* RUN THE KERNEL */
	/* THEN COPY THE RESULTS BACK */

	copy2_before= PAPI_get_real_usec();
	hipMemcpy(new_image.pixels, dev_new, (imageSize*sizeof(char)),hipMemcpyDeviceToHost);
	copy2_after= PAPI_get_real_usec();

	store_before=PAPI_get_real_usec();

	/* Write data back out to disk */
	store_jpeg("out.jpg",&new_image);

	store_after=PAPI_get_real_usec();

	/* Print timing results */
	printf("Load time: %lld\n",load_time-start_time);
        printf("Convolve time: %lld\n",convolve_time-load_time);
	printf("Copy host to device: %lld\n",(copy_after-copy_before));
        printf("Combine time: %lld\n",combine_after-combine_before);
	printf("Copy device to host: %lld\n",(copy2_after-copy2_before));
        printf("Store time: %lld\n",store_after-store_before);
	printf("Total time = %lld\n",store_after-start_time);

	return 0;
}
